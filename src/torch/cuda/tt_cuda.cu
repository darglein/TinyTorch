/**
 * Copyright (c) 2022 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */

#include "hip/hip_runtime.h"


namespace tinytorch
{
namespace cuda
{

static hipStream_t& thread_local_stream()
{
    static thread_local hipStream_t strm = 0;
    return strm;
}


hipStream_t getCurrentCUDAStream()
{
    return thread_local_stream();
}
void setCUDAStreamForThisThread(hipStream_t stream)
{
    thread_local_stream() = stream;
}

}  // namespace cuda
}  // namespace tinytorch