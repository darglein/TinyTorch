/**
 * Copyright (c) 2022 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */

#include "ops_impl_cuda_helper.h"
#include "tt_cuda.h"

#include <numeric>

namespace tinytorch
{
namespace cuda
{

static constexpr int MAX_DEVICES = 8;

static hipStream_t& thread_local_stream(int device_id)
{
    static thread_local hipStream_t strms[MAX_DEVICES] = {};
    return strms[device_id];
}

hipStream_t getCurrentCUDAStream()
{
    return thread_local_stream(getDevice());
}


hipStream_t getCUDAStream(Device device)
{
    CHECK_EQ(device.type(), kCUDA);
    return thread_local_stream(device.index());
}


void setCUDAStreamForThisThread(hipStream_t stream)
{
    thread_local_stream(getDevice()) = stream;
}

int getDevice()
{
    int device_index;
    CHECK_CUDA_ERROR(hipGetDevice(&device_index));
    CHECK_LT(device_index, MAX_DEVICES);
    return device_index;
}

void setDevice(int device_index)
{
    CHECK_CUDA_ERROR(hipSetDevice(device_index));
}

hipEvent_t getNextEvent()
{
    constexpr int MAX_EVENTS                                              = 128;
    static thread_local int current_event[MAX_DEVICES]                    = {};
    static thread_local hipEvent_t event_buffer[MAX_DEVICES][MAX_EVENTS] = {};

    int device   = getDevice();
    int& current = current_event[device];
    current      = (current + 1) % MAX_EVENTS;

    hipEvent_t& event = event_buffer[device][current];

    if (!event)
    {
        CHECK_CUDA_ERROR(hipEventCreate(&event));
    }

    return event;
}


std::vector<Device> GetCudaDevicesFromDeviceList(std::vector<int> device_list)
{
    int cuda_device_count;
    CHECK_CUDA_ERROR(hipGetDeviceCount(&cuda_device_count));

    if (cuda_device_count == 0)
    {
        throw std::runtime_error("No CUDA capable device found\n");
    }

    std::sort(device_list.begin(), device_list.end());
    device_list.erase(std::unique(device_list.begin(), device_list.end()), device_list.end());

    if (device_list.empty())
    {
        std::cout << "Parameter 'device_list' is empty. Defaulting to device 0\n";
        device_list.push_back(0);
    }
    if (device_list[0] == -1)
    {
        device_list.resize(cuda_device_count);
        std::iota(device_list.begin(), device_list.end(), 0);
    }

    std::vector<Device> result;
    for (int index : device_list)
    {
        if (index < 0)
        {
            throw std::runtime_error("Invalid negative device_id " + std::to_string(index) +
                                   ". Only allowed negative number is -1, in which case all GPUs are used.");
        }

        if (index < cuda_device_count)
        {
            result.push_back(Device(kCUDA, index));
        }
        else
        {
            std::cout << "Ignoring device id " << index << ". Only " << cuda_device_count << " GPUs are available.\n";
        }
    }

    if (result.empty())
    {
        throw std::runtime_error("No device id in 'device_list' matches a valid GPU id.");
    }



    return result;
}


}  // namespace cuda
}  // namespace tinytorch