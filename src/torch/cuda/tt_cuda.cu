/**
 * Copyright (c) 2022 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */

#include "tt_cuda.h"
#include "ops_impl_cuda_helper.h"

namespace tinytorch
{
namespace cuda
{

static hipStream_t& thread_local_stream()
{
	static thread_local hipStream_t strms[16] = {};
    return strms[getDevice()];
}

hipStream_t getCurrentCUDAStream()
{
    return thread_local_stream();
}

void setCUDAStreamForThisThread(hipStream_t stream)
{
    thread_local_stream() = stream;
}

int getDevice()
{
	int device_index;
	CHECK_CUDA_ERROR(hipGetDevice(&device_index));
	return device_index;
}

void setDevice(int device_index)
{
	CHECK_CUDA_ERROR(hipSetDevice(device_index));
}


}  // namespace cuda
}  // namespace tinytorch