#include "hip/hip_runtime.h"
#include "torch/core/ops.h"
#include "torch/core/tensor.h"

#include "torch/core/ops_functions.h"
#include "torch/core/ops_impl_shared.h"
#include "torch/core/tensor_info.h"
#include "torch/cuda/ops_operators_impl_cuda.h"
#include "torch/cuda/ops_impl_cuda_helper.h"

#include <hip/hip_runtime.h>

namespace tinytorch
{

template <typename T>
static void print_impl_cpu(std::ostream& strm, TensorInfo<T> a)
{
    for (int64_t i = 0; i < a.numel(); ++i)
    {
        strm << a[i] << " ";
    }
}

void print_impl_cuda(std::ostream& strm, Tensor t) 
{
    t = t.cpu();
    print_impl_cpu<float>(strm, t);
}

template <typename T>
__launch_bounds__(128) 
static __global__ void add_impl_cuda(TensorInfo<T> a, TensorInfo<T> b, TensorInfo<T> result)
{
    int64_t i = (int64_t)threadIdx.x + (int64_t)blockIdx.x * (int64_t)blockDim.x;
    if (i >= result.numel()) return;

    int64_t dims = result.dims;

    int64_t offset_a, offset_b;
    calculate_offsets(i, dims, a.sizes, b.sizes, a.strides, b.strides, offset_a, offset_b);
    result[i] = a.data[offset_a] + b.data[offset_b];
}

void add_impl_cuda(Tensor a, Tensor b, Tensor& result)
{
    SWITCH_MACRO_ALL(a.scalar_type(), a.numel(), add_impl_cuda, a, b, result);
}

template <typename T>
__launch_bounds__(128) 
static __global__ void add_impl_cuda(TensorInfo<T> a, double b, TensorInfo<T> result)
{
    int64_t i = (int64_t)threadIdx.x + (int64_t)blockIdx.x * (int64_t)blockDim.x;
    if (i >= result.numel()) return;

    result[i] = T(double(a[i]) + b);
}

void add_impl_cuda(Tensor a, double b, Tensor& result) 
{
    SWITCH_MACRO_ALL(a.scalar_type(), a.numel(), add_impl_cuda, a, b, result);
}

template <typename T>
__launch_bounds__(128) 
static __global__ void sub_impl_cuda(TensorInfo<T> a, TensorInfo<T> b, TensorInfo<T> result)
{
    int64_t i = (int64_t)threadIdx.x + (int64_t)blockIdx.x * (int64_t)blockDim.x;
    if (i >= result.numel()) return;

    int64_t dims = result.dims;

    int64_t offset_a, offset_b;
    calculate_offsets(i, dims, a.sizes, b.sizes, a.strides, b.strides, offset_a, offset_b);
    result[i] = a.data[offset_a] - b.data[offset_b];
}

void sub_impl_cuda(Tensor a, Tensor b, Tensor& result) 
{
    SWITCH_MACRO_ALL(a.scalar_type(), a.numel(), sub_impl_cuda, a, b, result);
}

template <typename T>
__launch_bounds__(128) static __global__ void sub_impl_cuda(TensorInfo<T> a, double b, TensorInfo<T> result)
{
    int64_t i = (int64_t)threadIdx.x + (int64_t)blockIdx.x * (int64_t)blockDim.x;
    if (i >= result.numel()) return;

    result[i] = T(double(a[i]) - b);
}

void sub_impl_cuda(Tensor a, double b, Tensor& result)
{
    SWITCH_MACRO_ALL(a.scalar_type(), a.numel(), sub_impl_cuda, a, b, result);
}

template <typename T>
__launch_bounds__(128) 
static __global__ void mult_impl_cuda(TensorInfo<T> a, TensorInfo<T> b, TensorInfo<T> result)
{
    int64_t i = (int64_t)threadIdx.x + (int64_t)blockIdx.x * (int64_t)blockDim.x;
    if (i >= result.numel()) return;

    int64_t dims = result.dims;

    int64_t offset_a, offset_b;
    calculate_offsets(i, dims, a.sizes, b.sizes, a.strides, b.strides, offset_a, offset_b);
    result[i] = a.data[offset_a] * b.data[offset_b];
}

void mult_impl_cuda(Tensor a, Tensor b, Tensor& result) 
{
    SWITCH_MACRO_ALL(a.scalar_type(), a.numel(), mult_impl_cuda, a, b, result);
}

template <typename T>
__launch_bounds__(128) 
static __global__ void mult_impl_cuda(TensorInfo<T> a, double b, TensorInfo<T> result)
{
    int64_t i = (int64_t)threadIdx.x + (int64_t)blockIdx.x * (int64_t)blockDim.x;
    if (i >= result.numel()) return;

    result[i] = T(double(a[i]) * b);
}

void mult_impl_cuda(Tensor a, double b, Tensor& result) 
{
    SWITCH_MACRO_ALL(a.scalar_type(), a.numel(), mult_impl_cuda, a, b, result);
}

template <typename T>
__launch_bounds__(128) 
static __global__ void div_impl_cuda(TensorInfo<T> a, TensorInfo<T> b, TensorInfo<T> result)
{
    int64_t i = (int64_t)threadIdx.x + (int64_t)blockIdx.x * (int64_t)blockDim.x;
    if (i >= result.numel()) return;

    int64_t dims = result.dims;

    int64_t offset_a, offset_b;
    calculate_offsets(i, dims, a.sizes, b.sizes, a.strides, b.strides, offset_a, offset_b);
    result[i] = a.data[offset_a] / b.data[offset_b];
}

void div_impl_cuda(Tensor a, Tensor b, Tensor& result) 
{
    SWITCH_MACRO_ALL(a.scalar_type(), a.numel(), div_impl_cuda, a, b, result);
}

template <typename T>
__launch_bounds__(128) 
static __global__ void div_impl_cuda(double a, TensorInfo<T> b, TensorInfo<T> result)
{
    int64_t i = (int64_t)threadIdx.x + (int64_t)blockIdx.x * (int64_t)blockDim.x;
    if (i >= result.numel()) return;

    result[i] = T(a / double(b[i]));
}

void div_impl_cuda(double a, Tensor b, Tensor& result) 
{
    SWITCH_MACRO_ALL(b.scalar_type(), b.numel(), div_impl_cuda, a, b, result);
}

template <typename T>
__launch_bounds__(128) 
static __global__ void equal_impl_cuda(TensorInfo<T> a, double b, TensorInfo<T> result)
{
    int64_t i = (int64_t)threadIdx.x + (int64_t)blockIdx.x * (int64_t)blockDim.x;
    if (i >= result.numel()) return;

    result[i] = T(a[i] == T(b));
}

void equal_impl_cuda(Tensor a, double b, Tensor& result) 
{
    SWITCH_MACRO_ALL(a.scalar_type(), a.numel(), equal_impl_cuda, a, b, result);
}

template <typename T>
__launch_bounds__(128) 
static __global__ void less_impl_cuda(TensorInfo<T> a, double b, TensorInfo<T> result)
{
    int64_t i = (int64_t)threadIdx.x + (int64_t)blockIdx.x * (int64_t)blockDim.x;
    if (i >= result.numel()) return;

    result[i] = T(a[i] < T(b));
}

void less_impl_cuda(Tensor a, double b, Tensor& result) 
{
    SWITCH_MACRO_ALL(a.scalar_type(), a.numel(), less_impl_cuda, a, b, result);
}

template <typename T>
__launch_bounds__(128) 
static __global__ void greater_impl_cuda(TensorInfo<T> a, double b, TensorInfo<T> result)
{
    int64_t i = (int64_t)threadIdx.x + (int64_t)blockIdx.x * (int64_t)blockDim.x;
    if (i >= result.numel()) return;

    result[i] = T(a[i] > T(b));
}

void greater_impl_cuda(Tensor a, double b, Tensor& result) 
{
    SWITCH_MACRO_ALL(a.scalar_type(), a.numel(), greater_impl_cuda, a, b, result);
}

}  // namespace tinytorch