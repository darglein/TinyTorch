/**
 * Copyright (c) 2022 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */

#include <map>
#include <mutex>

#include "cached_memory_allocator.h"
#include "torch/cuda/ops_impl_cuda_helper.h"
#include <hip/hip_runtime.h>


namespace tinytorch
{
namespace cuda
{

std::mutex mu;


static std::map<void*, int64_t> allocated_blocks;

static bool debug_print = true;

static void* malloc_async(int64_t size)
{
    std::unique_lock l(mu);
    void* ptr;
    auto strm       = cuda::getCurrentCUDAStream();
    auto cuda_error = hipMallocAsync(&ptr, size, strm);
    if (cuda_error == hipErrorOutOfMemory)
    {
        size_t mem_free, mem_total;
        hipMemGetInfo(&mem_free, &mem_total);
        CHECK_NE(cuda_error, hipErrorOutOfMemory)
            << " CUDA out of memory!\n"
            << "     Tried to allocate " << (size / 1000.0 / 1000.0) << "MB\n"
            << "     Free memory " << (mem_free / 1000.0 / 1000.0) << "MB\n"
            << "     Total memory " << (mem_total / 1000.0 / 1000.0) << "MB\n";
    }
    CHECK_CUDA_ERROR(cuda_error);
    CHECK_NOTNULL(ptr);
    allocated_blocks.insert({ptr, size});

    if (debug_print && (size / 1000.0 / 1000.0) > 100)
    {
        std::cout << "Allocate CUDA Memory: " << (size / 1000.0 / 1000.0) << "MB (" << ptr << ")\n";
    }

    return ptr;
}

static void* free_async(void* ptr)
{
    std::unique_lock l(mu);
    CHECK(allocated_blocks.find(ptr) != allocated_blocks.end());

    if (debug_print)
    {
        int64_t size = allocated_blocks[ptr];
        if ((size / 1000.0 / 1000.0) > 100)
        {
            std::cout << "Free CUDA Memory: " << (size / 1000.0 / 1000.0) << "MB (" << ptr << ")\n";
        }
    }

    allocated_blocks.erase(ptr);
    hipFreeAsync(ptr, cuda::getCurrentCUDAStream());
}

void* cuda_cached_malloc(int64_t size)
{
    if (size == 0)
    {
        return nullptr;
    }
    auto ptr = malloc_async(size);



    return ptr;
}
void cuda_cached_free(void* ptr)
{
    free_async(ptr);
}
void CUDACachingAllocator::emptyCache()
{
    // this frees unused values for the async allocator
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
}

}  // namespace cuda

}  // namespace tinytorch