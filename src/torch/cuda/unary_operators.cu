#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2022 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */

#include "torch/core/ops/ops_impl.h"
#include "torch/cuda/ops_impl_cuda.h"
#include "torch/cuda/ops_impl_cuda_helper.h"
#include "unary_operators.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>


namespace tinytorch
{

namespace cuda_impl
{

template <typename T, typename Op>
__launch_bounds__(128) static __global__
    void unary_operator_kernel(Op op, TensorInfoCuda<T> a, TensorInfoCuda<T> result)
{
    using G   = typename CpuComputeFloatType<T>::Type;
    int64_t i = (int64_t)threadIdx.x + (int64_t)blockIdx.x * (int64_t)blockDim.x;
    if (i >= result.numel()) return;
    {
        G input   = G(a[i]);
        G output  = op.forward(input);
        result[i] = T(output);
    }
}

template <typename T, typename Op>
__launch_bounds__(128) static __global__
    void unary_operator_backward_kernel(Op op, TensorInfoCuda<T> a, TensorInfoCuda<T> grad_a,
                                        TensorInfoCuda<T> grad_result)
{
    using G   = typename CpuComputeFloatType<T>::Type;
    int64_t i = (int64_t)threadIdx.x + (int64_t)blockIdx.x * (int64_t)blockDim.x;
    if (i >= grad_a.numel()) return;
    {
        G input       = G(a[i]);
        G grad_output = G(grad_result[i]);
        G grad_input  = op.backward(input, grad_output);
        grad_a[i]     = T(grad_input);
    }
}

#define SWITCH_MACRO_UNARY_OPERATOR(op, input, output)                                              \
    switch (input.scalar_type())                                                                    \
    {                                                                                               \
        CUDA_CASE_MACRO((unary_operator_kernel<uint8_t>), kUInt8, input.numel(), op, input, output) \
        CUDA_CASE_MACRO((unary_operator_kernel<int16_t>), kInt16, input.numel(), op, input, output) \
        CUDA_CASE_MACRO((unary_operator_kernel<int32_t>), kInt32, input.numel(), op, input, output) \
        CUDA_CASE_MACRO((unary_operator_kernel<int64_t>), kLong, input.numel(), op, input, output)  \
        CUDA_CASE_MACRO((unary_operator_kernel<__half>), kHalf, input.numel(), op, input, output)   \
        CUDA_CASE_MACRO((unary_operator_kernel<float>), kFloat, input.numel(), op, input, output)   \
        CUDA_CASE_MACRO((unary_operator_kernel<double>), kDouble, input.numel(), op, input, output) \
        default:                                                                                    \
            CHECK(false) << "invalid input type " << input.scalar_type();                           \
    }

#define SWITCH_MACRO_UNARY_OPERATOR_BACKWARD(op, input, grad_input, grad_result)                                 \
    switch (input.scalar_type())                                                                                 \
    {                                                                                                            \
        CUDA_CASE_MACRO((unary_operator_backward_kernel<uint8_t>), kUInt8, input.numel(), op, input, grad_input, \
                        grad_result)                                                                             \
        CUDA_CASE_MACRO((unary_operator_backward_kernel<int16_t>), kInt16, input.numel(), op, input, grad_input, \
                        grad_result)                                                                             \
        CUDA_CASE_MACRO((unary_operator_backward_kernel<int32_t>), kInt32, input.numel(), op, input, grad_input, \
                        grad_result)                                                                             \
        CUDA_CASE_MACRO((unary_operator_backward_kernel<int64_t>), kLong, input.numel(), op, input, grad_input,  \
                        grad_result)                                                                             \
        CUDA_CASE_MACRO((unary_operator_backward_kernel<__half>), kHalf, input.numel(), op, input, grad_input,   \
                        grad_result)                                                                             \
        CUDA_CASE_MACRO((unary_operator_backward_kernel<float>), kFloat, input.numel(), op, input, grad_input,   \
                        grad_result)                                                                             \
        CUDA_CASE_MACRO((unary_operator_backward_kernel<double>), kDouble, input.numel(), op, input, grad_input, \
                        grad_result)                                                                             \
        default:                                                                                                 \
            CHECK(false) << "invalid input type " << input.scalar_type();                                        \
    }
void abs_impl(Tensor a, Tensor result)
{
    SWITCH_MACRO_UNARY_OPERATOR(UnaryOperators::Abs(), a, result);
}
void round_impl(Tensor a, Tensor result)
{
    SWITCH_MACRO_UNARY_OPERATOR(UnaryOperators::Round(), a, result);
}
void sqrt_impl(Tensor a, Tensor result)
{
    SWITCH_MACRO_UNARY_OPERATOR(UnaryOperators::Sqrt(), a, result);
}
void log_impl(Tensor a, Tensor result)
{
    SWITCH_MACRO_UNARY_OPERATOR(UnaryOperators::Log(), a, result);
}
void exp_impl(Tensor a, Tensor result)
{
    SWITCH_MACRO_UNARY_OPERATOR(UnaryOperators::Exp(), a, result);
}
void sign_impl(Tensor a, Tensor result)
{
    SWITCH_MACRO_UNARY_OPERATOR(UnaryOperators::Sign(), a, result);
}
void sin_impl(Tensor a, Tensor result)
{
    SWITCH_MACRO_UNARY_OPERATOR(UnaryOperators::Sin(), a, result);
}
void cos_impl(Tensor a, Tensor result)
{
    SWITCH_MACRO_UNARY_OPERATOR(UnaryOperators::Cos(), a, result);
}
void relu_impl(Tensor a, Tensor result)
{
    SWITCH_MACRO_UNARY_OPERATOR(UnaryOperators::Relu(), a, result);
}
void sigmoid_impl(Tensor a, Tensor result)
{
    SWITCH_MACRO_UNARY_OPERATOR(UnaryOperators::Sigmoid(), a, result);
}
void sigmoid_backward_impl(Tensor a, Tensor grad_a, Tensor grad_result)
{
    SWITCH_MACRO_UNARY_OPERATOR_BACKWARD(UnaryOperators::Sigmoid(), a, grad_a, grad_result);
}

void softplus_impl(Tensor a, double beta, Tensor result)
{
    SWITCH_MACRO_UNARY_OPERATOR(UnaryOperators::Softplus(beta), a, result);
}
void softplus_backward_impl(Tensor a, double beta, Tensor grad_a, Tensor grad_result)
{
    SWITCH_MACRO_UNARY_OPERATOR_BACKWARD(UnaryOperators::Softplus(beta), a, grad_a, grad_result);
}
}  // namespace cuda_impl
}  // namespace tinytorch